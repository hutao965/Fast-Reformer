#include "hip/hip_runtime.h"
#include "kernels.cuh"

namespace FastReformer {

//TODO multi small fixed block size
// TODO add stream
// TODO fp16
// TODO template for common hidden size (1024 768 and so on)


// TODO half2 version
template<typename T, typename F>
__forceinline__ __device__ T reduce_warp(T value, F reduction) {
    #pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
        value = reduction(value, __shfl_xor_sync(0xffffffff, value, mask, 32));
    return value;
}

// instance for argmax
template<typename T, typename F>
__forceinline__ __device__ thrust::pair<T, int> reduce_warp(thrust::pair<T, int> value, F reduction) {
    #pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
        value = reduction(value, mask);
    return value;
}

template<typename T, typename F>
__forceinline__ __device__ T reduce_block(T value, F reduction, T null_value) {
    value = reduce_warp(value, reduction);
    __shared__ T s[32];
    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    if (laneId == 0) s[warpId] = value;
    __syncthreads();
    value = (laneId < ((blockDim.x + 31) >> 5)) ? s[laneId] : null_value;
    value = reduce_warp(value, reduction);
    return value;
}

template<typename T>
__forceinline__ __device__ T reduce_block_sum(T value) {
    return reduce_block(
        value,
        [](T x, T y){ return x + y; },
        static_cast<T>(0.0f)
    );
}

__forceinline__ __device__ float reduce_block_max(float value) {
    return reduce_block(
        value,
        [](float x, float y){ return max(x, y); },
        -FLT_MAX
    );
}

__forceinline__ __device__ thrust::pair<float, int> reduce_block_argmax(thrust::pair<float, int> value) {
    return reduce_block(
        value,
        [](thrust::pair<float, int> x, int mask){
            float other_first = __shfl_xor_sync(0xffffffff, x.first, mask, 32);
            int other_second = __shfl_xor_sync(0xffffffff, x.second, mask, 32);
            return x.first >= other_first ?
                   x : thrust::make_pair(other_first, other_second);
        },
        thrust::make_pair(-FLT_MAX, 0)
    );
}

// 1, 1, ..., 1 -> 1, 2, ..., 32
template<typename T>
__forceinline__ __device__ T reduce_warp_prefix_sum(T value, int laneId) {
    #pragma unroll
    for (int STRIDE = 1; STRIDE <= 16; STRIDE <<= 1) {
        T temp = __shfl_up_sync(0xffffffff, value, STRIDE, 32);
        value = laneId >= STRIDE ? temp + value : value;
    }
    return value;
}

// prefix sum for radix sort
// 1, 1, ..., 1 -> 0, 1, ..., 127
template<typename T>
__forceinline__ __device__ T reduce_block_prefix_sum(T value) {
    int warpId = threadIdx.x >> 5;
    int laneId = threadIdx.x & 31;
    value = reduce_warp_prefix_sum(value, laneId);
    // sum for each warp
    __shared__ T s[32];
    if (laneId == 31) s[warpId] = value;
    __syncthreads();
    if (warpId == 0) {
        T prefix = s[laneId];
        prefix = reduce_warp_prefix_sum(prefix, laneId);
        s[laneId] = prefix;
    }
    __syncthreads();
    value = __shfl_up_sync(0xffffffff, value, 1, 32);
    if (laneId == 0) value = static_cast<T>(0.0f);
    if (warpId != 0) value += s[warpId - 1];
    return value;
}


template<typename T>
__global__ void encoder_embedding(
    const int *input_ids, const T *tok_embd_weights,
    const T *pos_embd_weight_0, const T *pos_embd_weight_1,
    int pos_embds_dim_0, int pos_embds_dim_1, int pos_shape_0, int pos_shape_1,
    int hidden_size, int pad_id, int start_idx_pos_encodings,
    T *output, int *pad_mask) 
{
    int batch_seq_len = gridDim.x;
    int batch_seq_id = blockIdx.x;
    // int batch_size = gridDim.y;
    int batch_id = blockIdx.y;
    int pos_id = batch_seq_id + start_idx_pos_encodings;

    int input_id = input_ids[batch_id * batch_seq_len + batch_seq_id];
    pad_mask[batch_id * batch_seq_len + batch_seq_id] = static_cast<int>(input_id != pad_id);
    // tok embd
    T embd = tok_embd_weights[input_id * hidden_size + threadIdx.x];
    // axial pos embd
    embd += threadIdx.x < pos_embds_dim_0 ?
        pos_embd_weight_0[(pos_id / pos_shape_1) * pos_embds_dim_0 + threadIdx.x] :
        pos_embd_weight_1[(pos_id % pos_shape_1) * pos_embds_dim_1 + threadIdx.x - pos_embds_dim_0];
    // store
    output[
        batch_id * batch_seq_len * hidden_size +
        batch_seq_id * hidden_size +
        threadIdx.x
    ] = embd;
}

/**
 * @param input_ids [bs, seq_len]
 * @param tok_embd_weights [vocab_size, hidden_size]
 * @param pos_embd_weight_0 [pos_shape_0, 1, pos_embds_dim_0]
 * @param pos_embd_weight_1 [1, pos_shape_1, pos_embds_dim_1]
 * @param output [bs, seq_len, hidden_size]
 * @param pad_mask [bs, seq_len]
 *
 * gridDimX = batch_seq_len (dim x can be larger)
 * gridDimY = batch_size
 * blokcsize = hidden_size
 * 
 * only support 2 dim axial_pos_embds
 * only support hidden_size <= 1024
 */
template<typename T>
void encoder_embedding_launcher(
    const int *input_ids, const T *tok_embd_weights,
    const T *pos_embd_weight_0, const T *pos_embd_weight_1,
    int pos_embds_dim_0, int pos_embds_dim_1, int pos_shape_0, int pos_shape_1,
    int batch_size, int batch_seq_len, int hidden_size,
    int pad_id, int start_idx_pos_encodings,
    T *output, int *pad_mask) 
{
    assert(pos_embds_dim_0 % 32 == 0);
    assert(pos_embds_dim_1 % 32 == 0);
    assert(hidden_size <= 1024);
    encoder_embedding<<<dim3(batch_seq_len, batch_size), hidden_size>>>(
        input_ids, tok_embd_weights, pos_embd_weight_0, pos_embd_weight_1,
        pos_embds_dim_0, pos_embds_dim_1, pos_shape_0, pos_shape_1,
        hidden_size, pad_id, start_idx_pos_encodings,
        output, pad_mask);
}
template void encoder_embedding_launcher<float>(
    const int *input_ids, const float *tok_embd_weights,
    const float *pos_embd_weight_0, const float *pos_embd_weight_1,
    int pos_embds_dim_0, int pos_embds_dim_1, int pos_shape_0, int pos_shape_1,
    int batch_size, int batch_seq_len, int hidden_size,
    int pad_id, int start_idx_pos_encodings,
    float *output, int *pad_mask);


template<typename T>
__global__ void layer_norm(
    T *input, const T *weight, const T* bias,
    T eps)
{
    int norm_size = blockDim.x;
    T value = input[blockIdx.x * norm_size + threadIdx.x];
    T gamma = __ldg(&weight[threadIdx.x]);
    T beta = __ldg(&bias[threadIdx.x]);
    T mean = reduce_block_sum(value) / norm_size;
    T diff = value - mean;
    T var = diff * diff;
    var = reduce_block_sum(var) / norm_size;
    value = diff * rsqrtf(var + eps) * gamma + beta;
    input[blockIdx.x * norm_size + threadIdx.x] = value;
}
/**
 * @param input [size/norm_size, norm_size]
 * @param weight [norm_size]
 * @param bias [norm_size]
 */
// TODO only support norm_size <= 1024
template<typename T>
void layer_norm_launcher(
    T *input, const T *weight, const T *bias,
    T eps, int norm_size, int size)
{
    assert(norm_size <= 1024);
    layer_norm<<<size / norm_size, norm_size>>>(
        input, weight, bias, eps);
}
template void layer_norm_launcher<float>(
    float *input, const float *weight, const float *bias,
    float eps, int norm_size, int size);


template<typename T>
__global__ void bias_relu(T *input, const T *bias) {
    T value = input[blockIdx.x * blockDim.x + threadIdx.x] + __ldg(&bias[threadIdx.x]);
    value = max(value, static_cast<T>(0.0f));
    input[blockIdx.x * blockDim.x + threadIdx.x] = value;
}

/**
 * @param input [size/hidden_size, hidden_size]
 * @param bias [hidden_size]
 * @param output [size/hidden_size, hidden_size]
 */
template<typename T>
void bias_relu_launcher(
    T *input, const T *bias, int hidden_size, int size)
{
    bias_relu<<<size / hidden_size, hidden_size>>>(
        input, bias);
}
template void bias_relu_launcher<float>(
    float *input, const float *bias,
    int hidden_size, int size);


template<typename T>
__global__ void add_bias(T *input, const T *bias) {
    input[blockIdx.x * blockDim.x + threadIdx.x] += __ldg(&bias[threadIdx.x]);
}

template<typename T>
void add_bias_launcher(
    T *input, const T *bias, int hidden_size, int size)
{
    add_bias<<<size / hidden_size, hidden_size>>>(
        input, bias);
}
template void add_bias_launcher<float>(
    float *input, const float *bias,
    int hidden_size, int size);



template<typename T>
__global__ void softmax(T *input) {
    T value = input[blockIdx.x * blockDim.x + threadIdx.x];
    value -= reduce_block_max(value);
    value = expf(value);
    value /= reduce_block_sum(value);
    input[blockIdx.x * blockDim.x + threadIdx.x] = value;
}

template<typename T>
void softmax_launcher(
    T *input, int reduce_size, int size)
{
    softmax<<<size / reduce_size, reduce_size>>>(input);
}
template void softmax_launcher<float>(
    float *input, int reduce_size, int size);



template<typename T>
__global__ void atten_split_transpose(
    const T *input, int batch_size, int n_chunks,
    int chunk_len, int num_heads, int head_size,
    T *output)
{
    T value = input[threadIdx.x + blockIdx.x * blockDim.x];
    // 0, 1, 2, 3, 4 -> 0, 3, 1, 2, 4
    int idx_4 = threadIdx.x & (head_size - 1);
    int idx_3 = threadIdx.x / head_size;
    int idx_2 = blockIdx.x & (chunk_len - 1);
    int idx_1 = (blockIdx.x / chunk_len) & (n_chunks - 1);
    int idx_0 = blockIdx.x / chunk_len / n_chunks;
    output[
        idx_4 +
        head_size * (idx_2 +
        chunk_len * (idx_1 +
        n_chunks * (idx_3 +
        num_heads * idx_0)))
    ] = value;

}

// [bs, n_chunks, chunk_len, num_heads, head_size] ->
// [bs, num_heads, n_chunks, chunk_len, head_size]
template<typename T>
void atten_split_transpose_launcher(
    const T *input, int batch_size, int seq_len,
    int chunk_len, int num_heads, int head_size,
    T *output)
{
    atten_split_transpose<<<batch_size * seq_len, num_heads * head_size>>>(
        input, batch_size, seq_len/chunk_len, chunk_len, num_heads, head_size, output);
}
template void atten_split_transpose_launcher<float>(
    const float *input, int batch_size, int seq_len,
    int chunk_len, int num_heads, int head_size,
    float *output);



template<typename T>
__global__ void atten_merge_transpose(
    const T *input, int batch_size, int n_chunks,
    int chunk_len, int num_heads, int head_size,
    T *output)
{
    T value = input[threadIdx.x + blockIdx.x * blockDim.x];
    // 0, 1, 2, 3, 4 -> 0, 2, 3, 1, 4
    int idx_4 = threadIdx.x;
    int idx_3 = blockIdx.x & (chunk_len - 1);
    int idx_2 = (blockIdx.x / chunk_len) & (n_chunks - 1);
    int idx_1 = (blockIdx.x / chunk_len / n_chunks) & (num_heads - 1);
    int idx_0 = blockIdx.x / chunk_len / n_chunks / num_heads;
    output[
        idx_4 +
        head_size * (idx_1 +
        num_heads * (idx_3 +
        chunk_len * (idx_2 +
        n_chunks * idx_0)))
    ] = value;
}
// [bs, num_heads, n_chunks, chunk_len, head_size] ->
// [bs, n_chunks, chunk_len, num_heads, head_size]
template<typename T>
void atten_merge_transpose_launcher(
    const T *input, int batch_size, int seq_len,
    int chunk_len, int num_heads, int head_size,
    T *output)
{
    atten_merge_transpose<<<batch_size * num_heads * seq_len, head_size>>>(
        input, batch_size, seq_len/chunk_len, chunk_len, num_heads, head_size, output);
}
template void atten_merge_transpose_launcher<float>(
    const float *input, int batch_size, int seq_len,
    int chunk_len, int num_heads, int head_size,
    float *output);



template<typename T>
__global__ void look_adjacent(
    const T *input, int last_dim_size,
    int before, int after, int N,
    T *output)
{
    for (int k = 0; k < last_dim_size; k += blockDim.x) {
        T value = input[
            threadIdx.x + k +
            (blockIdx.x + blockIdx.y * gridDim.x) * last_dim_size
        ];
        // i on the N dim
        for (int i = 0; i < N; i ++) {
            int tgt_block_x = blockIdx.x - i + before;
            tgt_block_x = tgt_block_x < 0 ? tgt_block_x + gridDim.x : tgt_block_x;
            tgt_block_x = tgt_block_x >= gridDim.x ? tgt_block_x - gridDim.x : tgt_block_x;
            output[
                threadIdx.x + k +
                (i + (tgt_block_x + blockIdx.y * gridDim.x) * N) * last_dim_size
            ] = value;
        }
    }
}
// [bs * num_heads, n_chunks, chunk_len * head_size] ->
// [bs * num_heads, n_chunks, (Num_bef + Num_aft + 1), chunk_len * head_size]
// chunk_len * head_size = K * block_size
// gridDim.x = n_chunks
// gridDim.y = bs * num_heads
template<typename T>
void look_adjacent_launcher(
    const T *input, int batch_size, int num_heads, int n_chunks,
    int chunk_len, int head_size, int before, int after,
    T *output)
{
    // head_size can be 1, so should not use head_size as block_size
    int last_dim_size = chunk_len * head_size;
    int block_size = min(1024, last_dim_size);
    look_adjacent<<<dim3(n_chunks, batch_size * num_heads), block_size>>>(
        input, chunk_len * head_size, before, after, before + after + 1, output);
}
template void look_adjacent_launcher<int>(
    const int *input, int batch_size, int num_heads, int n_chunks,
    int chunk_len, int head_size, int before, int after,
    int *output);
template void look_adjacent_launcher<float>(
    const float *input, int batch_size, int num_heads, int n_chunks,
    int chunk_len, int head_size, int before, int after,
    float *output);



template<typename T>
__global__ void local_atten_enc_mask(
    T *qk_dots, const int *mask, T mask_value, int num_heads,
    int n_chunks, int chunk_len)
{
    int m = __ldg(&mask[
        threadIdx.x +
        (blockIdx.x / chunk_len + (blockIdx.y / num_heads) * n_chunks) * blockDim.x
    ]);

    int index = 
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
    qk_dots[index] = m ? qk_dots[index] : mask_value;

}

// [bs, num_heads, n_chunks, chunk_len,  N * chunk_len] * [bs, n_chunks, N * chunk_len]
// gridDim.x = n_chunks * chunk_len
// gridDim.y = batch_size * num_heads
// blockDim.x = N * chunk_len
template<typename T>
void local_atten_enc_mask_launcher(
    T *qk_dots, const int *mask, T mask_value, int batch_size, int num_heads,
    int n_chunks, int chunk_len, int N)
{
    local_atten_enc_mask<<<dim3(n_chunks * chunk_len, batch_size * num_heads), N * chunk_len>>>(
        qk_dots, mask, mask_value, num_heads, n_chunks, chunk_len);
}
template void local_atten_enc_mask_launcher<float>(
    float *qk_dots, const int *mask, float mask_value, int batch_size, int num_heads,
    int n_chunks, int chunk_len, int N);


template<typename T>
__global__ void repeat(const T *in, T *out) {
    out[
        threadIdx.x +
        (blockIdx.x + (blockIdx.y + blockIdx.z * gridDim.y) * gridDim.x) * blockDim.x
    ] =
    __ldg(&in[
        threadIdx.x +
        (blockIdx.x + blockIdx.z * gridDim.x) * blockDim.x
    ]);
}

/**
 * @param in [dim0, dim1]
 * @param out [dim0, repeat_num, dim1]
 */
template<typename T>
void repeat_launcher(
    const T *in, int dim0, int dim1, int repeat_num,
    T *out)
{
    // TODO if dim1 very small
    // [dim0, dim1/blocksize, blocksize] ->
    // [dim0, repeat_num, dim1/blocksize, blocksize]
    int blocksize = min(1024, dim1);
    dim3 grid(dim1/blocksize, repeat_num, dim0);
    repeat<<<grid, blocksize>>>(in, out);
}
template void repeat_launcher<float>(
    const float *in, int dim0, int dim1, int repeat_num,
    float *out);



/**
 * @param in [bs * num_heads * num_hashes * seq_len, num_bucket/2]
 * @param out [bs * num_heads * num_hashes * seq_len]
 */
template<typename T>
__global__ void lsh_bucket_argmax(
    const T *in, int *out)
{
    // argmax
    auto p = thrust::make_pair(
        in[threadIdx.x + blockIdx.x * blockDim.x],
        threadIdx.x
    );
    p = p.first > 0 ? p : thrust::make_pair(-p.first, p.second + blockDim.x);
    p = reduce_block_argmax(p);
    if (threadIdx.x == 0) {
        out[blockIdx.x] = p.second;
    }
}

/**
 * @param in [bs, num_heads, num_hashes, seq_len]
 * @param atten_mask [bs, seq_len]
 */
__global__ void lsh_bucket_mask_offset(
    int *in, const int *atten_mask, int num_bucket,
    int num_heads, int num_hashes)
{
    int value = in[threadIdx.x + blockIdx.x * blockDim.x];
    // mask
    int mask = __ldg(&atten_mask[threadIdx.x + (blockIdx.x / num_heads / num_hashes) * blockDim.x]);
    value = mask ? value : num_bucket - 1;
    //offset
    int offset = (blockIdx.x % num_hashes) * num_bucket;
    value += offset;
    in[threadIdx.x + blockIdx.x * blockDim.x] = value;
}


template<typename T>
void lsh_bucket_argmax_mask_offset_launcher(
    const T *in, const int *atten_mask,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int num_bucket,
    int *out)
{
    lsh_bucket_argmax<<<batch_size * num_heads * num_hashes * seq_len, num_bucket/2>>>(
        in, out);
    lsh_bucket_mask_offset<<<batch_size * num_heads * num_hashes, seq_len>>>(
        out, atten_mask, num_bucket + 1, num_heads, num_hashes);
}
template void lsh_bucket_argmax_mask_offset_launcher<float>(
    const float *in, const int *atten_mask,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int num_bucket,
    int *out);


// __global__ void arrange_last(int *out) {
//     out[
//         threadIdx.x +
//         (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
//     ] = threadIdx.x + blockIdx.x * blockDim.x;
// }

// /**
//  * @param out [size/last_size, last_size]
//  */
// void arrange_last_launcher(
//     int *out, int last_size, int size)
// {
//     // [size/last_size, last_size/blocksize, blocksize]
//     int blocksize = min(1024, last_size);
//     dim3 grid(last_size/blocksize, size/last_size);
//     arrange_last<<<grid, blocksize>>>(out);
// }



template<typename T, int ITEMS_PER_THREAD,
         int BLOCK_SIZE=128, int BEGIN_BIT=0, int END_BIT=sizeof(T)*8, int RADIX_BITS=4>
__global__ void block_unsigned_radix_sort(T *in, int *idx) {
    // linear load, line->ITEM_PER_THREAD, row->BLOCK_SIZE
    // 0, 1, ..., items_per_thread-1,
    // items_per_thread, ..., 2*items_per_thread-1,
    // ...
    T keys[ITEMS_PER_THREAD];
    int values[ITEMS_PER_THREAD];
    __shared__ T smem_keys[ITEMS_PER_THREAD * BLOCK_SIZE];
    __shared__ int smem_values[ITEMS_PER_THREAD * BLOCK_SIZE];

    // load
    #pragma unroll
    for (int i = 0; i < ITEMS_PER_THREAD; i++) {
        keys[i] = in[threadIdx.x * ITEMS_PER_THREAD + i + blockIdx.x * ITEMS_PER_THREAD * BLOCK_SIZE];
        values[i] = threadIdx.x * ITEMS_PER_THREAD + i;
    }

    #pragma unroll
    for (int start = BEGIN_BIT; start < END_BIT; start += RADIX_BITS) {
        int pass_bits = RADIX_BITS < END_BIT - start ? RADIX_BITS : END_BIT - start;
        // shift extractor
        auto digit_extractor = [start, pass_bits](T k) -> uint32_t {
            return uint32_t(k >> T(start)) & uint32_t((1 << pass_bits) - 1);
        };
        
        // rank keys
        // ranks = prev_bucket_ranks (ranks of prev buckets) +
        //         prev_lines_ranks (ranks of this bucket and prev lines) + 
        //         line_ranks (ranks of this bucket and this line)
        int bucket_counter[1<<RADIX_BITS] {};
        int ranks[ITEMS_PER_THREAD] {};
        uint32_t digits[ITEMS_PER_THREAD] {};

        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
            digits[ITEM] = digit_extractor(keys[ITEM]);
            // add line_ranks,
            ranks[ITEM] = bucket_counter[digits[ITEM]];
            bucket_counter[digits[ITEM]] += 1;
        }

        int bucket_prefix_sum_accum = 0;
        #pragma unroll
        for (int BIT = 0; BIT < (1<<RADIX_BITS); BIT++) {
            int prev_lines_rank = reduce_block_prefix_sum(bucket_counter[BIT]);
            int prev_bucket_rank = bucket_prefix_sum_accum;
            bucket_prefix_sum_accum += reduce_block_sum(bucket_counter[BIT]);
            // prev_lines_rank and prev_bucket_rank
            bucket_counter[BIT] = prev_lines_rank + prev_bucket_rank;
        }

        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
            ranks[ITEM] += bucket_counter[digits[ITEM]];
        }

        // scatter to smem
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++) {
            smem_keys[ranks[ITEM]] = keys[ITEM];
            smem_values[ranks[ITEM]] = values[ITEM];
        }
        __syncthreads();
        #pragma unroll
        for (int i = 0; i < ITEMS_PER_THREAD; i++) {
            keys[i] = smem_keys[threadIdx.x * ITEMS_PER_THREAD + i];
            values[i] = smem_values[threadIdx.x * ITEMS_PER_THREAD + i];
        }
    }

    #pragma unroll
    for (int i = 0; i < ITEMS_PER_THREAD; i++) {
        in[threadIdx.x * ITEMS_PER_THREAD + i + blockIdx.x * ITEMS_PER_THREAD * BLOCK_SIZE] = keys[i];
        idx[threadIdx.x * ITEMS_PER_THREAD + i + blockIdx.x * ITEMS_PER_THREAD * BLOCK_SIZE] = values[i];
    }
}

template<typename T>
void block_unsigned_radix_sort_launcher(T *in, int *idx, int grid, int N) {
    constexpr int BLOCK_SIZE = 128;
    switch(N) {
        case 128:
            block_unsigned_radix_sort<T, 128/BLOCK_SIZE><<<grid, BLOCK_SIZE>>>(in, idx);
            break;
        case 256:
            block_unsigned_radix_sort<T, 256/BLOCK_SIZE><<<grid, BLOCK_SIZE>>>(in, idx);
            break;
        case 512:
            block_unsigned_radix_sort<T, 512/BLOCK_SIZE><<<grid, BLOCK_SIZE>>>(in, idx);
            break;
        case 1024:
            block_unsigned_radix_sort<T, 1024/BLOCK_SIZE><<<grid, BLOCK_SIZE>>>(in, idx);
            break;
        case 2048:
            block_unsigned_radix_sort<T, 2048/BLOCK_SIZE><<<grid, BLOCK_SIZE>>>(in, idx);
            break;
        default:
            throw "num_hashes * seq_len must be 128, 256, 512, 1024 or 2048";
    }
}
template void block_unsigned_radix_sort_launcher<int>(int *in, int *idx, int grid, int N);



__global__ void lsh_scatter_undo_idx(
    int *sorted_idx, int *undo_sorted_idx, int seq_len)
{
    int idx = sorted_idx[
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
    ];
    // arange num_hashes*seq_len
    int value = threadIdx.x + blockIdx.x * blockDim.x;

    undo_sorted_idx[
        idx +
        blockIdx.y * gridDim.x * blockDim.x] = value;

    sorted_idx[
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
    ] = idx % seq_len;
}

/**
 * produce undo_sorted_idx, and also make sorted_idx %= seq_len
 * @param sorted_idx [bs*num_heads, num_hashes*seq_len]
 * @param undo_sorted_idx [bs*num_heads, num_hashes*seq_len]
 */
void lsh_scatter_undo_idx_launcher(
    int *sorted_idx, int *undo_sorted_idx,
    int batch_size, int num_heads, int num_hashes, int seq_len)
{
    // [bs*num_heads, num_hashes*seq_len/blocksize, blocksize]
    int blocksize = min(1024, num_hashes * seq_len);
    dim3 grid(num_hashes * seq_len / blocksize, batch_size * num_heads);
    lsh_scatter_undo_idx<<<grid, blocksize>>>(
        sorted_idx, undo_sorted_idx, seq_len);
}


template<typename T>
__global__ void lsh_gather_by_expansion(
    const T *in, const int *idx, int seq_len, T *out)
{
    int gather_idx = idx[blockIdx.x + blockIdx.y * gridDim.x];
    out[
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
    ] = 
    in[
        threadIdx.x +
        (gather_idx % seq_len + blockIdx.y * seq_len) * blockDim.x
    ];
}
/**
 * gather the num_hashes*seq_len dim
 * @param in [bs, num_heads, seq_len, head_size] (expand to [bs, num_heads, num_hashes*seq_len, head_size])
 * @param idx [bs, num_heads, num_hashes*seq_len]
 * @param out [bs, num_heads, num_hashes*seq_len, head_size]
 */
template<typename T>
void lsh_gather_by_expansion_launcher(
    const T *in, const int *idx, int batch_size,
    int num_heads, int num_hashes, int seq_len, int head_size,
    T *out)
{
    dim3 grid(num_hashes * seq_len, batch_size * num_heads);
    lsh_gather_by_expansion<<<grid, head_size>>>(
        in, idx, seq_len, out);
}
template void lsh_gather_by_expansion_launcher<float>(
    const float *in, const int *idx, int batch_size,
    int num_heads, int num_hashes, int seq_len, int head_size,
    float *out);


template<typename T>
__global__ void lsh_len_norm(
    const T *in, T norm_scalar, T *out)
{
    T value = in[threadIdx.x + blockIdx.x * blockDim.x];
    T rstd = rsqrtf(reduce_block_sum(value * value) / blockDim.x + static_cast<T>(1e-6f));
    out[threadIdx.x + blockIdx.x * blockDim.x] = value * rstd * norm_scalar;
}

template<typename T>
void lsh_len_norm_launcher(
    const T *in, int norm_size, int size, T norm_scalar, T *out)
{
    lsh_len_norm<<<size/norm_size, norm_size>>>(in, norm_scalar, out);
}
template void lsh_len_norm_launcher<float>(
    const float *in, int norm_size, int size, float norm_scalar, float *out);


/**
 * atten mask and self mask
 * gridDim.x = bs * num_heads * num_hashes*seq_len/chunk_len * chunk_len
 * blockDim.x = N * chunk_len
 * 
 * @param qk_dots [bs, num_heads, num_hashes*seq_len/chunk_len, chunk_len,  N * chunk_len]
 * @param q_idx [bs, num_heads, num_hashes*seq_len/chunk_len, chunk_len]
 * @param k_idx [bs, num_heads, num_hashes*seq_len/chunk_len, N * chunk_len]
 * @param atten_mask [bs, seq_len]
 */
template<typename T>
__global__ void lsh_enc_mask(
    T *qk_dots, const int *q_idx, const int *k_idx, const int *atten_mask,
    T mask_value, T self_mask_value, int num_heads, int num_hashes,
    int seq_len, int chunk_len)
{
    T value = qk_dots[threadIdx.x + blockIdx.x * blockDim.x];
    // mask
    int k_gather_idx = __ldg(&k_idx[
        threadIdx.x +
        (blockIdx.x / chunk_len) * blockDim.x
    ]);
    int mask = __ldg(&atten_mask[
        k_gather_idx +
        (blockIdx.x / (num_heads * num_hashes * seq_len)) * seq_len
    ]);
    // self mask
    int q = __ldg(&q_idx[
        blockIdx.x
    ]);
    int k = __ldg(&k_idx[
        threadIdx.x +
        (blockIdx.x / chunk_len) * blockDim.x
    ]);
    bool self_mask = q != k;

    value = mask ? value : mask_value;
    value = self_mask ? value : self_mask_value;
    qk_dots[threadIdx.x + blockIdx.x * blockDim.x] = value;
}

template<typename T>
void lsh_enc_mask_launcher(
    T *qk_dots, const int *q_idx, const int *k_idx, const int *atten_mask,
    T mask_value, T self_mask_value, int batch_size, int num_heads, int num_hashes,
    int seq_len, int chunk_len, int N)
{
    dim3 grid(batch_size * num_heads * num_hashes * seq_len);
    lsh_enc_mask<<<grid, N * chunk_len>>>(
        qk_dots, q_idx, k_idx, atten_mask, mask_value, self_mask_value,
        num_heads, num_hashes, seq_len, chunk_len);
}

template void lsh_enc_mask_launcher<float>(
    float *qk_dots, const int *q_idx, const int *k_idx, const int *atten_mask,
    float mask_value, float self_mask_value, int batch_size, int num_heads, int num_hashes,
    int seq_len, int chunk_len, int N);


/**
 * softmax version that also return logits
 */
template<typename T>
__global__ void softmax_with_logits(T *input, T *logits) {
    T value = input[blockIdx.x * blockDim.x + threadIdx.x];
    T max_value = reduce_block_max(value);
    value -= max_value;
    value = expf(value);
    T sum_value = reduce_block_sum(value);
    input[blockIdx.x * blockDim.x + threadIdx.x] = value / sum_value;
    if (threadIdx.x == 0) {
        logits[blockIdx.x] = max_value + logf(sum_value);
    }
}

template<typename T>
void softmax_with_logits_launcher(
    T *input, T *logits, int reduce_size, int size)
{
    softmax_with_logits<<<size / reduce_size, reduce_size>>>(input, logits);
}
template void softmax_with_logits_launcher<float>(
    float *input, float *logits, int reduce_size, int size);


template<typename T>
__global__ void lsh_undo_sort(
    const int *undo_sort_idx, const T *vec, const T *logits,
    T *rev_vec, T *rev_logits)
{
    int idx = undo_sort_idx[blockIdx.x + blockIdx.y * gridDim.x];
    rev_vec[
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
    ] =
    vec[
        threadIdx.x +
        (idx + blockIdx.y * gridDim.x) * blockDim.x
    ];

    if (threadIdx.x == 0) {
        rev_logits[blockIdx.x + blockIdx.y * gridDim.x] = logits[idx + blockIdx.y * gridDim.x];
    }
}

/**
 * gather
 * @param undo_sort_idx [bs*num_heads, num_hashes*seq_len]
 * @param vec [bs*num_heads, num_hashes*seq_len, head_size]
 * @param logits [bs*num_heads, num_hashes*seq_len]
 * @param rev_vec [bs*num_heads, num_hashes*seq_len, head_size]
 * @param rev_logits [bs*num_heads, num_hashes*seq_len]
 */
template<typename T>
void lsh_undo_sort_launcher(
    const int *undo_sort_idx, const T *vec, const T *logits,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int head_size,
    T *rev_vec, T *rev_logits)
{
    dim3 grid(num_hashes * seq_len, batch_size * num_heads);
    lsh_undo_sort<<<grid, head_size>>>(
        undo_sort_idx, vec, logits, rev_vec, rev_logits);
}
template void lsh_undo_sort_launcher<float>(
    const int *undo_sort_idx, const float *vec, const float *logits,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int head_size,
    float *rev_vec, float *rev_logits);


/**
 * gridDim.x = seq_len
 * gridDim.y = bs * num_heads
 * blockDim.x = head_size
 * 
 * @param in [bs, num_heads, num_hashes, seq_len, head_size]
 * @param logits [bs, num_heads, num_hashes, seq_len]
 * @param out [bs, num_heads, seq_len, head_size]
 */
template<typename T, int num_hashes>
__global__ void sum_up_hashes(
    const T *in, const T *logits, T *out)
{
    T vs[num_hashes];
    T ls[num_hashes];
    # pragma unroll
    for (int i = 0; i < num_hashes; i ++) {
        int logits_idx = 
            blockIdx.x +
            (i + blockIdx.y * num_hashes) * gridDim.x;
        vs[i] = in[threadIdx.x + logits_idx * blockDim.x];
        ls[i] = __ldg(&logits[logits_idx]);
    }
    T logsumexp = static_cast<T>(0.0f);
    T *max_l = thrust::max_element(thrust::device, ls, ls + num_hashes);
    # pragma unroll
    for (int i = 0; i < num_hashes; i ++) {
        logsumexp += expf(ls[i] - *max_l);
    }
    logsumexp = logf(logsumexp) + *max_l;
    T res = static_cast<T>(0.0f);
    # pragma unroll
    for (int i = 0; i < num_hashes; i ++) {
        res += vs[i] * expf(ls[i] - logsumexp);
    }
    out[
        threadIdx.x +
        (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
    ] = res;
}

template<typename T>
void sum_up_hashes_launcher(
    const T *in, const T *logits,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int head_size,
    T *out)
{
    dim3 grid(seq_len, batch_size * num_heads);
    dim3 block(head_size);
    switch(num_hashes) {
        case 1:
            thrust::copy(
                thrust::device,
                in, in + batch_size * num_heads * seq_len * head_size,
                out);
            break;
        case 2:
            sum_up_hashes<T, 2><<<grid, block>>>(in, logits, out);
            break;
        case 4:
            sum_up_hashes<T, 4><<<grid, block>>>(in, logits, out);
            break;
        case 8:
            sum_up_hashes<T, 8><<<grid, block>>>(in, logits, out);
            break;
        default:
            throw "num_hashes must be 1, 2, 4 or 8";
    }
}
template void sum_up_hashes_launcher<float>(
    const float *in, const float *logits,
    int batch_size, int num_heads, int num_hashes,
    int seq_len, int head_size,
    float *out);


template<typename T>
__global__ void add(T *first, T *second) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    T sum = first[idx] + second[idx];
    first[idx] = sum;
    second[idx] = sum;
}
/**
 * first and second both store the result
 */
template<typename T>
void add_launcher(T *first, T *second, int size) {
    int blocksize = min(1024, size);
    add<<<size/blocksize, blocksize>>>(first, second);
}
template void add_launcher<float>(float *first, float *second, int size);


} // namespace FastReformer